/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/hip/sampling.hpp>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_bf16.h>
#include "flashinfer_ops.h"
// #include "pytorch_extension_utils.h"

using namespace flashinfer;

#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")

#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)

#define CHECK_DIM(d, x) TORCH_CHECK(x.dim() == d, #x " must be a " #d "D tensor")

#define CHECK_SHAPE(a, b) check_shape(a, b, #a, #b)

#define CHECK_EQ(a, b) TORCH_CHECK((a) == (b), "CHECK_EQ(" #a ", " #b ") failed. ", a, " vs ", b)

#define CHECK_GE(a, b) TORCH_CHECK((a) >= (b), "CHECK_GE(" #a ", " #b ") failed. ", a, " vs ", b)

torch::Tensor sampling_from_probs(torch::Tensor probs, torch::Tensor uniform_samples,
                                  bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(1, uniform_samples);  // uniform_samples: (batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(0));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));

  hipError_t status = sampling::SamplingFromProb(static_cast<float*>(probs.data_ptr()),
                                                  static_cast<float*>(uniform_samples.data_ptr()),
                                                  static_cast<int*>(samples.data_ptr()), batch_size,
                                                  vocab_size, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "SamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));
  return samples;
}

std::vector<torch::Tensor> top_p_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     std::optional<torch::Tensor> maybe_top_p_arr,
                                                     double top_p_val, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_p_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_p_rounds = uniform_samples.size(0);
  bool has_top_p_arr = maybe_top_p_arr.has_value();
  auto top_p_arr = maybe_top_p_arr.value_or(torch::empty({0}, torch::dtype(torch::kFloat32)));
  if (has_top_p_arr) {
    CHECK_INPUT(top_p_arr);
    CHECK_DIM(1, top_p_arr);  // top_p_arr: (batch_size,)
    CHECK_EQ(top_p_arr.size(0), batch_size);
    CHECK_EQ(top_p_arr.device(), device);
  }
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  top_p_arr = top_p_arr.to(torch::kFloat32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopPSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()),
      has_top_p_arr ? static_cast<float*>(top_p_arr.data_ptr()) : nullptr, batch_size, top_p_val,
      vocab_size, max_top_p_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopPSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

std::vector<torch::Tensor> top_k_sampling_from_probs(torch::Tensor probs,
                                                     torch::Tensor uniform_samples,
                                                     std::optional<torch::Tensor> maybe_top_k_arr,
                                                     unsigned int top_k_val, bool deterministic) {
  CHECK_INPUT(probs);
  CHECK_INPUT(uniform_samples);
  auto device = probs.device();
  CHECK_EQ(uniform_samples.device(), device);
  CHECK_DIM(2, probs);            // probs: (batch_size, vocab_size)
  CHECK_DIM(2, uniform_samples);  // uniform_samples: (max_top_k_rounds, batch_size)
  CHECK_EQ(probs.size(0), uniform_samples.size(1));
  unsigned int batch_size = probs.size(0);
  unsigned int vocab_size = probs.size(1);
  unsigned int max_top_k_rounds = uniform_samples.size(0);
  bool has_top_k_arr = maybe_top_k_arr.has_value();
  auto top_k_arr = maybe_top_k_arr.value_or(torch::empty({0}, torch::dtype(torch::kInt32)));
  if (has_top_k_arr) {
    CHECK_INPUT(top_k_arr);
    CHECK_DIM(1, top_k_arr);  // top_k_arr: (batch_size,)
    CHECK_EQ(top_k_arr.size(0), batch_size);
    CHECK_EQ(top_k_arr.device(), device);
  }
  probs = probs.to(torch::kFloat32);
  uniform_samples = uniform_samples.to(torch::kFloat32);
  top_k_arr = top_k_arr.to(torch::kInt32);

  hipStream_t torch_current_stream = c10::cuda::getCurrentCUDAStream(device.index());
  auto samples = torch::empty({batch_size}, torch::dtype(torch::kInt32).device(device));
  auto success = torch::empty({batch_size}, torch::dtype(torch::kBool).device(device));

  hipError_t status = sampling::TopKSamplingFromProb<float, int>(
      static_cast<float*>(probs.data_ptr()), static_cast<float*>(uniform_samples.data_ptr()),
      static_cast<int*>(samples.data_ptr()), static_cast<bool*>(success.data_ptr()),
      has_top_k_arr ? static_cast<float*>(top_k_arr.data_ptr()) : nullptr, batch_size, top_k_val,
      vocab_size, max_top_k_rounds, deterministic, torch_current_stream);
  TORCH_CHECK(status == hipSuccess, "TopKSamplingFromProbs failed with error code " +
                                         std::string(hipGetErrorString(status)));

  return {samples, success};
}

